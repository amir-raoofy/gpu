#include "db.cuh"
int main( int argc,char ** argv){

	// argument handling
	if (argc<5){
		fprintf(stderr, "usage: ./sim_cuda <maximum_number_of_threads> <number_of_particles> <simulation_time> <time_step>\n");
		exit(1);
	}
		
	// set the simulation parameters
	const int max_thread = atoi(argv[1]);
	const int N = atoi(argv[2]);	//number of particles
	int blocks = int(N/max_thread)+1;
	int T =atoi(argv[3]) ;		// duration of the simulation
	float dt =atoi(argv[4]);		//time steps
	
	//declare input and output array on the Host
	Particle h_particles[N];
	initial_condition(h_particles, N);
	Particle h_output[N];	
	
	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles, N * sizeof(Particle));
	hipMalloc((void **) &d_output,    N * sizeof(Particle));		
	
	//Transfer arrays to the Device
	hipMemcpy(d_particles, h_particles, N * sizeof(Particle), hipMemcpyHostToDevice);
	
	// run the kernel with N threads and 1 Blocks
	update_position<<<blocks,N>>>(dt,T,N,d_particles,d_output,max_thread);
	
	//write the solution back to the Host
	hipMemcpy(h_output, d_output, N * sizeof(Particle), hipMemcpyDeviceToHost);
	
/*	for(int i = 0 ; i < N ; i ++){
		std::cout<<h_output[i].get_position()[0]<<","<<h_output[i].get_position()[1]\
		<<std::endl;
	}
*/	
	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
