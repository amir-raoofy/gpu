#include "hip/hip_runtime.h"
#include "db.cuh"


// implementation of the constructor of the particles

__host__ Particle::Particle(){
	
	this ->m    = 1  ;
	this ->q    = 1  ;
	this ->x[0] = 0.5;
	this ->x[1] = 0.5;
	this ->v[0] = 0.5;
	this ->v[1] = 0.5;
	this ->E[0] = 0.0;
	this ->E[1] = 0.0;

};

__device__ __host__ double Particle::get_mass(){
	return this ->m;
};

__device__ __host__ double Particle::get_charge(){
	return this ->q;
};

__device__ __host__ double * Particle::get_position(){
	return this ->x;
};

__device__ __host__ double * Particle::get_velocity(){
	return this ->v;
};

__device__ __host__ double * Particle::get_interaction(){
	return this-> I;
};

__device__ __host__ void Particle::set_position(double* position){
	this->x[0]=position[0];
	this->x[1]=position[1];
};

__device__ __host__ void Particle::set_velocity(double* velocity){
	this->v[0]=velocity[0];
	this->v[1]=velocity[1];
};
__device__ __host__ void Particle::update_field(int N, int index, Particle * particles){
	
	this->set_interaction(index, N, particles);
	this->set_field();

};

__device__ __host__ void Particle::solve_time_step(double dt){

	v[0] = v[0] + dt * q * E[0] / m ;
	x[0] = x[0] + dt * v [0];
	
	v[1] = v[1] + dt * q * E[1] / m ;
	x[1] = x[1] + dt * v [1];
};

__device__ __host__ void Particle::set_field(){
	electricField(this->E);
};

__device__ __host__ void Particle::set_interaction(int N, int index, Particle * particles){
	
	double x_1 = this->x[0];
	double y_1 = this->x[1];
	for(int i = 0 ; i < index ; i++){
		double x_2 = particles[i].get_position()[0];
		double y_2 = particles[i].get_position()[1];
		double q_2 = particles[i].get_charge();
		double r_2 = (x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2);
		double r_3_2 = sqrtf(r_2*r_2*r_2);
//		this->I[0] += q*(x_1-x_2)/r_3_2;
//		this->I[1] += q*(y_1-y_2)/r_3_2;
		this->I[0] += 0;
		this->I[1] += 0;
	}
	for(int i=index + 1;i < N ; i++){
		double x_2 = particles[i].get_position()[0];
		double y_2 = particles[i].get_position()[1];
		double q_2 = particles[i].get_charge();
		double r_2 = (x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2);
		double r_3_2 = sqrtf(r_2*r_2*r_2);
//		this->I[0] += q*(x_1-x_2)/r_3_2;
//		this->I[1] += q*(y_1-y_2)/r_3_2;
		this->I[0] += 0;
		this->I[1] += 0;
	}
}

Simulation::Simulation(double dt, double T, int N, int max_threads, int blocks, int output_flag, Particle* particles_host_in, Particle * particles_host_out):
	_dt(dt), _T(T), _N(N), _max_threads(max_threads), _blocks(blocks), _output_flag(output_flag),
	_particles_host_in(particles_host_in), _particles_host_out(particles_host_out)
	{}
		
void Simulation::solve(){
	
	//initialize the positions and the veolicities of the partilces for simulation	
	initial_condition(_particles_host_in, _N);
	
	//declare arrays which will be transfered to the Device
	Particle * particles_device_in;
	Particle * particles_device_out;

	//allocate memory space on the Device
	hipMalloc((void **) &particles_device_in,  _N * sizeof(Particle));
	hipMalloc((void **) &particles_device_out, _N * sizeof(Particle));		
	
	//Transfer arrays to the Device
	hipMemcpy(particles_device_in, _particles_host_in, _N * sizeof(Particle), hipMemcpyHostToDevice);

	// run the kernel with N threads and 1 Blocks
	for( int i = 0; i < int(_T/_dt)+1; i++){		
	    update_position<<<_blocks,((_N<_max_threads)?_N:_max_threads)>>>(_dt, _T, _N, particles_device_in, particles_device_out, _max_threads);
	    hipDeviceSynchronize(); 
	}
	//write the solution back to the Host
	hipMemcpy(_particles_host_out, particles_device_out, _N * sizeof(Particle), hipMemcpyDeviceToHost);

	hipFree(particles_device_in );
	hipFree(particles_device_out);

}

__global__ void update_position(double dt, double T, const int N,\
 Particle * particles, Particle * d_output,const int max_thread){
	
	// define the index of the thread
	int t_index = threadIdx.x;
	int b_index = blockIdx.x;
	int index = (b_index*max_thread)+t_index;
	if (index >= N)
	    return
	
	// find the position of the index-th particle at time T
	particles[index].update_field(N, index , particles);
	particles[index].solve_time_step(dt);
	
	//write back the updated particles into the output
	d_output[index] = particles[index];
};

__host__ __device__ void electricField(double* E){
	E[0]=0.0;
	E[1]=0.0;
}

__host__ void initial_condition(Particle * particles,int N){
	double pos[2];
	for( int i = 0; i < N; i++){
		pos[0]=(double)(rand() % 1000)/100;
		pos[1]=(double)(rand() % 1000)/100;
		particles[i].set_position(pos);
	}
}
