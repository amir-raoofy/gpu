#include "db.cuh"
int main( int argc,char ** argv){
	
	// set the simulation parameters
	const int max_thread = 1024;
	const int N = 2049;	//number of particles
	int blocks;
	if(N % max_thread == 0){
		blocks = int(N/max_thread);
	}
	else{
		blocks = int(N/max_thread)+1;
	}
	
	int T = 0;		// duration of the simulation
	float dt = 0.1;		//time steps
	
	
	
	//declare input and output array on the Host
	Particle h_particles[N];
	initial_condition(h_particles, N);
	Particle h_output[N];	
	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes = N * sizeof(Particle);
	int output_array_bytes = particles_array_bytes;

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		
	
	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);
	
	// run the kernel with N threads and 1 Blocks
	for( int i = 0; i < int(T/dt)+1; i++){
	update_position<<<blocks,((N<max_thread)?N:max_thread)>>>(dt,T,N,d_particles,d_output,max_thread,blocks);
	}
	
	//write the solution back to the Host
	hipMemcpy(h_output,d_output,output_array_bytes,hipMemcpyDeviceToHost);
	
	for(int i = 0 ; i < N ; i ++){
		std::cout<<h_output[i].get_position()[0]<<","<<h_output[i].get_position()[1]\
		<<std::endl;
	}
	//std::cout<<positions_array_bytes<<std::endl;
	
	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
