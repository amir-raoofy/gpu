#include "db.cuh"

int main( int argc,char ** argv){
	
	// set the simulation parameters
	const int max_thread	= 1024;		//maximum number of threads per block
	const int N				= NUMBER;	//number of particles
	const int T				= 1000;		// duration of the simulation
	const float dt 			= 0.1;		//time steps	

	
	//declare input and output array on the Host
	Particle h_particles[N];
	
	//set particle position randomly 
	initial_condition(h_particles,N);
	
	//output array on the Host
	float h_output_x[N];
	float h_output_y[N];
	
	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes	= N * sizeof(Particle);
	int output_array_bytes 		= N * sizeof(float);

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	float * d_output_x;
	float * d_output_y;
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output_x,output_array_bytes);		
	hipMalloc((void **) &d_output_y,output_array_bytes);	
			
	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);
		
	// run the kernel with N threads and 1 Blocks
	std::ofstream myfile;
	myfile.open("data.txt");
	for(int i = 0; i < int(T/dt); i++){
		update_position<<<1,N>>>(dt,T,N,d_particles,d_output_x,d_output_y,max_thread);
		if(i%10 == 1){
			hipMemcpy(h_output_x,d_output_x,output_array_bytes,hipMemcpyDeviceToHost);
			hipMemcpy(h_output_y,d_output_y,output_array_bytes,hipMemcpyDeviceToHost);
			for(int j = 0; j < N; j++){
				myfile << h_output_x[j]<<","<<h_output_y[j]<<'\t';
			}
			myfile << std::endl;
		}
	}
	myfile.close();

	
	//write the solution back to the Host
	hipMemcpy(h_output_x,d_output_x,output_array_bytes,hipMemcpyDeviceToHost);
	hipMemcpy(h_output_y,d_output_y,output_array_bytes,hipMemcpyDeviceToHost);
	
	for(int i = 0 ; i < N ; i++){
		std::cout<<h_output_x[i]<<","<<h_output_y[i]\
		<<std::endl;
	}
	//std::cout<<positions_array_bytes<<std::endl;
	
	hipFree(d_particles);
	hipFree(d_output_x);
	hipFree(d_output_y);
	return 0;
}
