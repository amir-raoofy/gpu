#include "db.cuh"
#include "output.h"

int main(){
	
	// set the simulation parameters
	const int max_thread	= 512;		//maximum number of threads per block
	const int N				= NUMBER;	//number of particles
	const int T				= 1000;		// duration of the simulation
	const float dt 			= 0.1;		//time steps	

	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes	= N * sizeof(Particle);
	int output_array_bytes 		= N * sizeof(Particle);
	
	
	//declare input and output array on the Host which are the same
	Particle h_particles[N];
	//set particle position randomly 
	initial_condition(h_particles,N);
	

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;


	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		

			
	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);
		
	
	//std::ofstream myfile;
	//myfile.open("data");
	//myfile << "{";
	//int count = 0;

	Output *output =new Output(N,h_particles);

	for(int i = 0; i < int(T/dt); i++){
		
		// run the kernel with N threads and 1 Blocks
		update_position<<<1,NUMBER>>>(dt,T,N,d_particles,d_output,max_thread);

		hipMemcpy(h_particles,d_output,particles_array_bytes,hipMemcpyDeviceToHost);
		output->setTimeStep(i);
		output->writeFile();

		//hipDeviceSynchronize();
	/*	if(i%10 == 0){
			hipMemcpy(h_particles,d_output,particles_array_bytes,hipMemcpyDeviceToHost);
			for(int j = 0; j < N ; j++ ){
				if( i/10 != (int(T/dt)/10)-1 || j !=N-1){
					myfile <<"{"<< h_particles[j].get_position()[0]<<","<<h_particles[j].get_position()[1]<<","<<count<<"},";
				}
				else{
				myfile <<"{"<< h_particles[j].get_position()[0]<<","<<h_particles[j].get_position()[1]<<","<<count<<"}";
				}
			}
		count++;
		}*/
	}
	
	
	//myfile << "}";
	//myfile.close();

	
	//write the solution back to the Host
	hipMemcpy(h_particles,d_output,particles_array_bytes,hipMemcpyDeviceToHost);


/*	
	for(int i = N-10 ; i < N ; i++){

		std::cout<<h_particles[i].get_position()[0]<<","<<h_particles[i].get_position()[1]\
		<<std::endl;
	}
	std::cout<<"size of:= "<<particles_array_bytes<<std::endl;
*/
	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
