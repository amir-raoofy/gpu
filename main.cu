#include "db.cuh"
int main( int argc,char ** argv){
	
	// set the simulation parameters
	const int max_thread = 1024;
	const int N = 100;	//number of particles
	int blocks;
	if(N % max_thread == 0){
		blocks = int(N/max_thread);
	}
	else{
		blocks = int(N/max_thread)+1;
	}
	
	int T = 10;		// duration of the simulation
	float dt = 0.001;		//time steps
	
	
	
	//declare input and output array on the Host
	Particle h_particles[N];
	initial_condition(h_particles, N);
	/*double dummy[2];
	dummy[0] = 3;
	dummy[1] = 3;
	h_particles[0].set_position(dummy);
	dummy[1] = 5;
	dummy[0] = 5;
//	h_particles[0].set_interaction(2,0,h_particles);
	h_particles[1].set_position(dummy);*/
	Particle h_output[N];

	//std::cout<< h_particles[0].get_interaction()[1]<<std::endl;	
	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes = N * sizeof(Particle);
	int output_array_bytes = particles_array_bytes;

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		
	
	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);
	
	// run the kernel with N threads and 1 Blocks
	for( int i = 0; i < int(T/dt); i++){
	update_position<<<blocks,((N<max_thread)?N:max_thread)>>>(dt,T,N,d_particles,d_output,max_thread,blocks);
	hipDeviceSynchronize();
	}
	
	//write the solution back to the Host
	hipMemcpy(h_output,d_output,output_array_bytes,hipMemcpyDeviceToHost);
	
	for(int i = 0 ; i < N ; i ++){
		std::cout<<h_output[i].get_position()[0]<<","<<h_output[i].get_position()[1]\
		<<std::endl;
	}
	//std::cout<<positions_array_bytes<<std::endl;
	
	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
