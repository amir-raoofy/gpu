#include "hip/hip_runtime.h"
#include "db.cuh"


// implementation of the constructor of the particles

__host__ Particle::Particle(){
	
	this ->m    = 1  ;
	this ->q    = 0.001;
	this ->x[0] = 0.0;
	this ->x[1] = 0.0;
	this ->v[0] = 0.0;
	this ->v[1] = 0.0;
	this ->E[0] = 0.0;
	this ->E[1] = 0.0;
	this ->I[0] = 0.0;
	this ->I[1] = 0.0;

};

__device__ __host__ float Particle::get_mass(){
	return this ->m;
};
__device__ __host__ float Particle::get_charge(){
	return this ->q;
};
__device__ __host__ float * Particle::get_position(){
	return this ->x;
};
__device__ __host__ float * Particle::get_velocity(){
	return this ->v;
};

__device__ __host__ float * Particle::get_interaction(){
	return this-> I;
};

__device__ __host__ void Particle::set_position(float* position){
	this->x[0]=position[0];
	this->x[1]=position[1];
};
__device__ __host__ void Particle::set_velocity(float* velocity){
	this->v[0]=velocity[0];
	this->v[1]=velocity[1];
};
__device__ void Particle::update_field(int N, int index, Particle * particles){
	
	this->set_interaction(N, index, particles);

	this->set_field();


};

//solve the newton equation by euler method
__device__ __host__ void Particle::solve_time_step(float dt){

	v[0] = v[0] + dt * q * (E[0]+I[0]) / m ;
	x[0] = x[0] + dt * v [0];
	
	v[1] = v[1] + dt * q * (E[1]+I[1]) / m ;
	x[1] = x[1] + dt * v [1];
};

//set the electronic field of each particle
__device__ __host__ void Particle::set_field(){
	electricField(this->E,this->x);
};

//calculate the interaction between the particle[index] and other particles
__device__ void Particle::set_interaction(int N, int index, Particle * particles){
	
	float x_1 = this->x[0];
	float y_1 = this->x[1];
	float x_2 ;
	float y_2 ;
	float r_sqrt;
	float r_3_2;
	float q_2;
	
	this->I[0]=0;
	this->I[1]=0;
	for(int i = 0 ; i < index ; i++){
		x_2 = particles[i].get_position()[0];

		y_2 = particles[i].get_position()[1];

		q_2 = particles[i].get_charge();

		r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
		r_3_2 = r_sqrt*r_sqrt*r_sqrt;
		this->I[0] += q_2*(x_1-x_2)/r_3_2;

		this->I[1] += q_2*(y_1-y_2)/r_3_2;

	}
	for(int i=index + 1;i < N ; i++){
		x_2 = particles[i].get_position()[0];

		y_2 = particles[i].get_position()[1];

		q_2 = particles[i].get_charge();
		r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
		r_3_2 = r_sqrt*r_sqrt*r_sqrt;
		this->I[0] += q_2*(x_1-x_2)/r_3_2;
		this->I[1] += q_2*(y_1-y_2)/r_3_2;

	}
}
		

__global__ void update_position(float dt, float T, const int N,\
 Particle * particles, Particle * d_output, const int max_thread){

	int index = threadIdx.x;
	
	//allocate shared memory and copy the particles into it
	//__shared__ Particle sh_particles[NUMBER];
	//sh_particles[index] = particles[index];
	//__syncthreads();
	
	//update field and solve for a time step
	particles[index].update_field(N, index, particles);
	__syncthreads();

	particles[index].solve_time_step(dt);
	__syncthreads();
	
	//particles[index] = sh_particles[index];
	//__syncthreads();
	
	d_output[index] = particles[index];
	__syncthreads();
	//__threadfence();
		
	
	
};

//the electrical field acts similar to a infinite wall
__host__ __device__ void electricField(float* E, float* x){
	E[0]=1/x[0] + 1/(x[0] - 100000);
	E[0]*=E[0];
	E[1]=1/x[1] + 1/(x[1] - 100000);
	E[1]*=E[1];
}

__host__ void initial_condition(Particle * particles,int N){
	float pos[2];
	for( int i = 0; i < N; i++){
		pos[0]=2500+(float)(rand()%100000) / 20.0;
		pos[1]=2500+(float)(rand()%100000) / 20.0;
		particles[i].set_position(pos);
	}
}
	
	
	
