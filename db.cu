#include "hip/hip_runtime.h"
#include "db.cuh"


// implementation of the constructor of the particles

__host__ Particle::Particle(){
	
	this ->m    = 1  ;
	this ->q    = 0.000001  ;
	this ->x[0] = 0.0;
	this ->x[1] = 0.0;
	this ->v[0] = 0.0;
	this ->v[1] = 0.0;
	this ->E[0] = 0.0;
	this ->E[1] = 0.0;
	this ->I[0] = 0.0;
	this ->I[1] = 0.0;

};

__device__ __host__ double Particle::get_mass(){
	return this ->m;
};
__device__ __host__ double Particle::get_charge(){
	return this ->q;
};
__device__ __host__ double * Particle::get_position(){
	return this ->x;
};
__device__ __host__ double * Particle::get_velocity(){
	return this ->v;
};

__device__ __host__ double * Particle::get_interaction(){
	return this-> I;
};

__device__ __host__ void Particle::set_position(double* position){
	this->x[0]=position[0];
	this->x[1]=position[1];
};
__device__ __host__ void Particle::set_velocity(double* velocity){
	this->v[0]=velocity[0];
	this->v[1]=velocity[1];
};
__device__ __host__ void Particle::update_field(int N, int index, Particle * particles){
	
	this->set_interaction(N, index, particles);
	this->set_field();

};


__device__ __host__ void Particle::solve_time_step(double dt){

	v[0] = v[0] + dt * q * (E[0]+I[0]) / m ;
	x[0] = x[0] + dt * v [0];
	
	v[1] = v[1] + dt * q * (E[1]+I[1]) / m ;
	x[1] = x[1] + dt * v [1];
};

__device__ __host__ void Particle::set_field(){
	electricField(this->E,this->x);
};


__device__ __host__ void Particle::set_interaction(int N, int index, Particle * particles){
	
	double x_1 = this->x[0];
	double y_1 = this->x[1];
	double x_2 ;
	double y_2 ;
	double r_sqrt;
	double r_3_2;
	double q_2;
	
	this->I[0]=0;
	this->I[1]=0;
	for(int i = 0 ; i < index ; i++){
		x_2 = particles[i].get_position()[0];
		y_2 = particles[i].get_position()[1];
		q_2 = particles[i].get_charge();
		r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
		r_3_2 = r_sqrt*r_sqrt*r_sqrt;
		this->I[0] += q_2*(x_1-x_2)/r_3_2;
		this->I[1] += q_2*(y_1-y_2)/r_3_2;
//		this->I[0] = 1;
//		this->I[1] = 0;
	}
	for(int i=index + 1;i < N ; i++){
		x_2 = particles[i].get_position()[0];
		y_2 = particles[i].get_position()[1];
		q_2 = particles[i].get_charge();
		r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
		r_3_2 = r_sqrt*r_sqrt*r_sqrt;
		this->I[0] += q_2*(x_1-x_2)/r_3_2;
		this->I[1] += q_2*(y_1-y_2)/r_3_2;
//		this->I[0] = 1;
//		this->I[1] = 0;
	}
}
		

__global__ void update_position(double dt, double T, const int N,\
 Particle * particles, Particle * d_output,const int max_thread,int blocks){
	
	// define the index of the thread
	int t_index = threadIdx.x;
	int b_index = blockIdx.x;
	int index = (b_index*max_thread)+t_index;
	if(index >= N)
		return;
	// find the position of the index-th particle at time T

	particles[index].update_field(N, index , particles);
	particles[index].solve_time_step(dt);
	
	
	//write back the updated particles into the output
	d_output[index] = particles[index];
};


__host__ __device__ void electricField(double* E, double* x){
	E[0]=1/x[0] + 1/(x[0] - 1000);
	E[0]*=E[0];
	E[1]=1/x[1] + 1/(x[1] - 1000);
	E[1]*=E[1];
}

__host__ void initial_condition(Particle * particles,int N){
	double pos[2];
	//double velo[2];
	for( int i = 0; i < N; i++){
		pos[0]=250+(double)(rand()%100000) / 200.0;
		pos[1]=250+(double)(rand()%100000) / 200.0;
		//velo[0]=25+(double)(rand()%10000) / 200.0;
		//velo[1]=25+(double)(rand()%10000) / 200.0;
		particles[i].set_position(pos);
	}
}
	
	
	
