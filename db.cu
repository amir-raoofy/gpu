#include "hip/hip_runtime.h"
#include "db.cuh"


// implementation of the constructor of the particles

Particle::Particle(){
	
	this ->m    = 1  ;
	this ->q    = 1  ;
	this ->x[0] = 0.5;
	this ->x[1] = 0.5;
	this ->v[0] = 0.5;
	this ->v[1] = 0.5;
	this ->E[0] = 0;
	this ->E[1] = 0;

};

__device__ __host__ double Particle::get_mass(){
	return this ->m;
};
__device__ __host__ double Particle::get_charge(){
	return this ->q;
};
__device__ __host__ double * Particle::get_position(){
	return this ->x;
};
__device__ __host__ double * Particle::get_velocity(){
	return this ->v;
};

__device__ __host__ void Particle::set_position(double* position){
	this->x[0]=position[0];
	this->x[1]=position[1];
};
__device__ __host__ void Particle::set_velocity(double* velocity){
	this->v[0]=velocity[0];
	this->v[1]=velocity[1];
};

__device__ __host__ void ElectricField(double* E, const Particle& P){
	
};

__host__ __device__ void diff_solve (double dt, double T, int N, Particle *particles){
	int Nt=T/dt;
	for (int i=0; i<Nt; i++){
		for (int n=0; n<N; n++){
			// update the electric field
			particles[n].set_field();
			particles[n].solve_time_step(dt);
		}
	}
};

__device__ __host__ void Particle::solve_time_step(double dt){

	v[0] = v[0] + dt * q * E[0] / m ;
	x[0] = x[0] + dt * v [0];
	
	v[1] = v[1] + dt * q * E[1] / m ;
	x[1] = x[1] + dt * v [1];
};
__device__ __host__ void Particle::set_field(){
	ElectricField(this->E, *this);
};

__global__ void solve(double dt, double T, const int N, Particle * particles, Particle * d_output){
	
	// define the index of the thread
	int index = threadIdx.x;
	
	//write particles into the shared memory
	__shared__ Particle sh_particles[10];
	sh_particles[index] = particles[index]; // copying entire position into the shared memory
	
	//synchronize the thread
	__syncthreads();
	
	for( int i = 0; i < int(T/dt); i++){
		
		particles[index].solve_time_step(dt); 
	}
	d_output[index] = particles[index];
};


	
