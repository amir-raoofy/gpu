#include "hip/hip_runtime.h"
#include "db.cuh"


// implementation of the constructor of the particles

__host__ Particle::Particle(){
	
        this ->m    = 1  ;
        this ->q    = 0.000001  ;
        this ->x[0] = 0.0;
        this ->x[1] = 0.0;
        this ->v[0] = 0.0;
        this ->v[1] = 0.0;
        this ->E[0] = 0.0;
        this ->E[1] = 0.0;
        this ->I[0] = 0.0;
        this ->I[1] = 0.0;
};

__device__ __host__ double Particle::get_mass(){
	return this ->m;
};

__device__ __host__ double Particle::get_charge(){
	return this ->q;
};

__device__ __host__ double * Particle::get_position(){
	return this ->x;
};

__device__ __host__ double * Particle::get_velocity(){
	return this ->v;
};

__device__ __host__ double * Particle::get_interaction(){
	return this-> I;
};

__device__ __host__ void Particle::set_position(double* position){
	this->x[0]=position[0];
	this->x[1]=position[1];
};

__device__ __host__ void Particle::set_velocity(double* velocity){
	this->v[0]=velocity[0];
	this->v[1]=velocity[1];
};
__device__ __host__ void Particle::update_field(int N, int index, Particle * particles){
	
	this->set_interaction(N, index, particles);
	this->set_field();

};

__device__ __host__ void Particle::solve_time_step(double dt){

        v[0] = v[0] + dt * q * (E[0]+I[0]) / m ;
        x[0] = x[0] + dt * v [0];

        v[1] = v[1] + dt * q * (E[1]+I[1]) / m ;
        x[1] = x[1] + dt * v [1];
};

__device__ __host__ void Particle::set_field(){
	electricField(this->E,this->x);
};

__device__ __host__ void Particle::set_interaction(int N, int index, Particle * particles){

        double x_1 = this->x[0];
        double y_1 = this->x[1];
        double x_2 ;
        double y_2 ;
        double r_sqrt;
        double r_3_2;
        double q_2;

        this->I[0]=0;
        this->I[1]=0;
        for(int i = 0 ; i < index ; i++){
                x_2 = particles[i].get_position()[0];
                y_2 = particles[i].get_position()[1];
                q_2 = particles[i].get_charge();
                r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
                r_3_2 = r_sqrt*r_sqrt*r_sqrt;
                this->I[0] += q_2*(x_1-x_2)/r_3_2;
                this->I[1] += q_2*(y_1-y_2)/r_3_2;
//              this->I[0] = 1;
//              this->I[1] = 0;
        }
        for(int i=index + 1;i < N ; i++){
                x_2 = particles[i].get_position()[0];
                y_2 = particles[i].get_position()[1];
                q_2 = particles[i].get_charge();
                r_sqrt = sqrt(sqrt((x_1-x_2)*(x_1-x_2) + (y_1-y_2)*(y_1-y_2)));
                r_3_2 = r_sqrt*r_sqrt*r_sqrt;
                this->I[0] += q_2*(x_1-x_2)/r_3_2;
                this->I[1] += q_2*(y_1-y_2)/r_3_2;
//              this->I[0] = 1;
//              this->I[1] = 0;
        }
};

Simulation::Simulation(double dt, double T, int N, int max_threads, int blocks, int output_flag, Particle* particles_host_in, Particle * particles_host_out):
	_dt(dt), _T(T), _N(N), _max_threads(max_threads), _blocks(blocks), _output_flag(output_flag),
	_particles_host_in(particles_host_in), _particles_host_out(particles_host_out)
	{}
		
void Simulation::solve(){
	
	//initialize the positions and the veolicities of the partilces for simulation	
	initial_condition(_particles_host_in, _N);
	
	//declare arrays which will be transfered to the Device
	Particle * particles_device_in;
	Particle * particles_device_out;

	//allocate memory space on the Device
	hipMalloc((void **) &particles_device_in,  _N * sizeof(Particle));
	hipMalloc((void **) &particles_device_out, _N * sizeof(Particle));		
	
	//Transfer arrays to the Device
	hipMemcpy(particles_device_in, _particles_host_in, _N * sizeof(Particle), hipMemcpyHostToDevice);

	// run the kernel with N threads and 1 Blocks
	for( int i = 0; i < int(_T/_dt)+1; i++){		
	    update_position<<<_blocks,((_N<_max_threads)?_N:_max_threads)>>>(_dt, _T, _N, particles_device_in, particles_device_out, _max_threads,_blocks);
	    hipDeviceSynchronize(); 
	}
	//write the solution back to the Host
	hipMemcpy(_particles_host_out, particles_device_out, _N * sizeof(Particle), hipMemcpyDeviceToHost);

	hipFree(particles_device_in );
	hipFree(particles_device_out);

};


__global__ void update_position(double dt, double T, const int N,\
 Particle * particles, Particle * d_output,const int max_thread,int blocks){

        // define the index of the thread
        int t_index = threadIdx.x;
        int b_index = blockIdx.x;
        int index = (b_index*max_thread)+t_index;
        if(index >= N)
                return;
        // find the position of the index-th particle at time T

        particles[index].update_field(N, index , particles);
        particles[index].solve_time_step(dt);


        //write back the updated particles into the output
        d_output[index] = particles[index];
};

__host__ __device__ void electricField(double* E, double* x){
        E[0]=1/x[0] + 1/(x[0] - 1000);
        E[0]*=E[0];
        E[1]=1/x[1] + 1/(x[1] - 1000);
        E[1]*=E[1];
};

__host__ void initial_condition(Particle * particles,int N){
        double pos[2];
        //double velo[2];
        for( int i = 0; i < N; i++){
                pos[0]=250+(double)(rand()%100000) / 200.0;
                pos[1]=250+(double)(rand()%100000) / 200.0;
                //velo[0]=25+(double)(rand()%10000) / 200.0;
                //velo[1]=25+(double)(rand()%10000) / 200.0;
                particles[i].set_position(pos);
        }
};
