#include "db.cuh"
#include "output.cuh"

int main (int argc,char ** argv){
	
	// set the simulation parameters
	Parameters * parameters = new Parameters (argc, argv);
	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes	= parameters->_N * sizeof(Particle);
	int output_array_bytes 		= parameters->_N * sizeof(Particle);
	
	//declare input and output array on the Host which are the same
	Particle h_particles[parameters->_N];
	//set particle position randomly 
	initial_condition(h_particles,parameters);
	

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	Output *output =new Output(parameters->_N,h_particles,parameters->_output_flag);
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		

	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);

	for(int i = 0; i < int(parameters->_T/parameters->_dt); i++){
		
		// run the kernel with N threads and 1 Blocks
		update_position<<<1,(parameters->_N)>>>(parameters->_dt,parameters->_T,parameters->_N,d_particles,d_output,parameters->_max_threads);
		//hipDeviceSynchronize();
		
		if (parameters->_output_flag){
			if (!(i%10))
				hipMemcpy(h_particles,d_output,particles_array_bytes,hipMemcpyDeviceToHost);
			output->setTimeStep(i);
			output->writeFile();
		}
	}

	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
