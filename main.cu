#include "db.cuh"
int main( int argc,char ** argv){
	
	// set the simulation parameters
	
	const int N = 10;	//number of particles
	int T = 1;		// duration of the simulation
	float dt = 0.1;		//time steps
	
	
	
	//declare input and output array on the Host
	Particle h_particles[N];
	Particle h_output[N];	
	
	//calculate the size of the arrays to be allocated
	int particles_array_bytes = N * sizeof(Particle);
	int output_array_bytes = particles_array_bytes;

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		
	
	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);
	
	// run the kernel with N threads and 1 Blocks
	solve<<<1,N>>>(dt,T,N,d_particles,d_output);
	
	//write the solution back to the Host
	hipMemcpy(h_output,d_output,output_array_bytes,hipMemcpyDeviceToHost);
	
	for(int i = 0 ; i < N ; i ++){
		std::cout<<h_output[i].get_position()[0]<<","<<h_output[i].get_position()[1]\
		<<std::endl;
	}
	//std::cout<<positions_array_bytes<<std::endl;
	
	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
