#include "db.cuh"
#include "output.cuh"

int main(){
	
	// set the simulation parameters
	const int max_thread	= 512;		//maximum number of threads per block
	const int N		= NUMBER;	//number of particles
	const int T		= 1000;		//duration of the simulation
	const float dt 		= 0.1;		//time steps	
	const int output_flag	= 0;		//

	//calculate the size of the arrays to be allocated
	int particles_array_bytes	= N * sizeof(Particle);
	int output_array_bytes 		= N * sizeof(Particle);
	
	//declare input and output array on the Host which are the same
	Particle h_particles[N];
	//set particle position randomly 
	initial_condition(h_particles,N);
	

	//declare arrays which will be transfered to the Device
	Particle * d_particles;
	Particle * d_output;
	
	Output *output =new Output(N,h_particles,output_flag);
	
	//allocate memory space on the Device
	hipMalloc((void **) &d_particles,particles_array_bytes);
	hipMalloc((void **) &d_output,output_array_bytes);		

	//Transfer arrays to the Device
	hipMemcpy(d_particles,h_particles,particles_array_bytes,hipMemcpyHostToDevice);

	for(int i = 0; i < int(T/dt); i++){
		
		// run the kernel with N threads and 1 Blocks
		update_position<<<1,NUMBER>>>(dt,T,N,d_particles,d_output,max_thread);
		//hipDeviceSynchronize();
		
		if (output_flag){
			if (!(i%10))
				hipMemcpy(h_particles,d_output,particles_array_bytes,hipMemcpyDeviceToHost);
			output->setTimeStep(i);
			output->writeFile();
		}
	}

	hipFree(d_particles);
	hipFree(d_output);
	
	return 0;
}
